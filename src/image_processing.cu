#include "hip/hip_runtime.h"
#include <filesystem>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include "../include/image_processing.cuh"
#include "../include/helper_image.h"

#define CH1_BLOCK_SIZE dim3(16, 16, 1)
#define CH1_WINDOW_SIZE dim3(4, 3, 1)

__device__ inline int x_gradient(u_char p11, u_char p12, u_char p13, u_char p21, u_char p22, u_char p23, u_char p31, u_char p32, u_char p33)
{
	return p11 + p21 + p31 - p13 - p23 - p33;
}

__device__ inline int y_gradient(u_char p11, u_char p12, u_char p13, u_char p21, u_char p22, u_char p23, u_char p31, u_char p32, u_char p33)
{
	return p11 + p12 + p13 - p31 - p32 - p33;
}

__device__ inline int prewitt_gradient(u_char p11, u_char p12, u_char p13, u_char p21, u_char p22, u_char p23, u_char p31, u_char p32, u_char p33)
{
	int gx = x_gradient(p11, p12, p13, p21, p22, p23, p31, p32, p33);
	int gy = y_gradient(p11, p12, p13, p21, p22, p23, p31, p32, p33);
	int sum = abs(gx) + abs(gy);
	sum = sum > 255 ? 255 : sum;
	sum = sum < 0 ? 0 : sum;
	return sum;
}

__global__ void prewitt_cylce_1ch(const u_char* data, u_char* res, const size_t width, const size_t height, const size_t pitch_in, const size_t pitch_out) 
{
	size_t offset_x = (CH1_WINDOW_SIZE.x - 2) * (blockIdx.x * blockDim.x + threadIdx.x);
	size_t offset_y = (CH1_WINDOW_SIZE.y - 2) * (blockIdx.y * blockDim.y + threadIdx.y);
	
	//can calculate at least 3x3
	if(offset_x + 2 < width && offset_y + 2 < height)
	{
		//hardcode for WINDOW_SIZE dim3(4, 3, 1);
		u_int32_t words[3];
		memcpy(words, data + offset_x + offset_y * pitch_in, sizeof(u_int32_t));
		memcpy(words + 1, data + offset_x + (offset_y + 1) * pitch_in, sizeof(u_int32_t));
		memcpy(words + 2, data + offset_x + (offset_y + 2) * pitch_in, sizeof(u_int32_t));

		u_char out[2];
		out[0] = prewitt_gradient(words[0] >> 24 & 0xFF, words[0] >> 16 & 0xFF, words[0] >> 8 & 0xFF, words[1] >> 24 & 0xFF, words[1] >> 16 & 0xFF, words[1] >> 8 & 0xFF, words[2] >> 24 & 0xFF, words[2] >> 16 & 0xFF, words[2] >> 8 & 0xFF);
		out[1] = prewitt_gradient(words[0] >> 16 & 0xFF, words[0] >> 8 & 0xFF, words[0] & 0xFF, words[1] >> 16 & 0xFF, words[1] >> 8 & 0xFF, words[1] & 0xFF, words[2] >> 16 & 0xFF, words[2] >> 8 & 0xFF, words[2] & 0xFF);
	
		memcpy(res + offset_x + 1 + (offset_y + 1) * pitch_out, out, 2 * sizeof(u_char));
	}
}

void prewittGPU(const std::string& file)
{
	u_char* device_data, * device_res;
	u_char* input = 0, * output;
	u_int width, height, channels;
	size_t pitch_in, pitch_out;

	__loadPPM(file.c_str(), &input, &width, &height, &channels);
	output = (u_char*)malloc(width * height * channels * sizeof(u_char));

	hipMallocPitch((void**)&device_data, &pitch_in, width * channels * sizeof(u_char), height);
	hipMallocPitch((void**)&device_res, &pitch_out, width * channels * sizeof(u_char), height);
	hipMemcpy2D(device_data, pitch_in, input, width * channels * sizeof(u_char), width * channels * sizeof(u_char), height, hipMemcpyHostToDevice);

	switch (channels)
	{
		case 1:
			{
				dim3 real_block(CH1_BLOCK_SIZE.x * (CH1_WINDOW_SIZE.x - 2), CH1_BLOCK_SIZE.y * (CH1_WINDOW_SIZE.y - 2));
				dim3 grid_size((width - 2 + real_block.x - 1) / real_block.x, (height - 2 + real_block.y - 1) / real_block.y);

				prewitt_cylce_1ch << <grid_size, CH1_BLOCK_SIZE >> > (device_data, device_res, width * channels, height, pitch_in, pitch_out);
				break;
			}
		case 3:
		default:
			break;
	}

	hipDeviceSynchronize();

	hipMemcpy2D(output, width * channels * sizeof(u_char), device_res, pitch_out, width * channels * sizeof(u_char), height, hipMemcpyDeviceToHost);
	hipFree(device_data);
	hipFree(device_res);

	hipHostFree(input);

	std::filesystem::path input_path(file);
	std::filesystem::path ouput_path(input_path.parent_path());
	ouput_path /= "result";
	ouput_path /= input_path.stem().string() + ".prewitt" + input_path.extension().string();

	__savePPM(ouput_path.c_str(), output, width, height, channels);
	free(output);
}